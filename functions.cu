
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

__device__ __hip_bfloat16 f(int x) {
  return __int2bfloat16_rd(x);
}

__device__ __hip_bfloat16 f(float x) {
  return __float2bfloat16(x);
}


__device__ __hip_bfloat16 add(__hip_bfloat16 x, __hip_bfloat16 y) {
  return __hadd(x, y);
}

__device__ __hip_bfloat16 mul(__hip_bfloat16 x, __hip_bfloat16 y) {
  return __hmul(x, y);
}


__device__ __hip_bfloat16 muladd(__hip_bfloat16 x, __hip_bfloat16 y, __hip_bfloat16 z) {
  return __hfma(x, y, z);
}
